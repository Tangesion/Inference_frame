#include "hip/hip_runtime.h"
#include<stdio.h>
#include<torch/types.h>



__global__ void flash_attn_kernel(const float *Q, const float *K, const float *V, float *O, int bc, int br, const int n, const int d, const int tc, const int tr,
    float *m, float *l, const float softmax_scale, float *s, float *out_q, float *out_k, float *out_o)
{
    int b = blockIdx.y;
    int h = blockIdx.x;
    int b_size = gridDim.y;
    int h_size = gridDim.x;
    
    // block.dimx = n * d

    int qkv_offset = b * h_size * n * d + h * n * d;
    int lm_offset = b * h_size * n + h * n;
    int s_offset = b * h_size * n * n + h * n * n;

    extern  __shared__ float sram[];
    //div sram to q,k,v,s
    float *sq = sram;
    float *sk = sq + br * d;
    float *sv = sk + bc * d;
    float *S = sv + bc * d;
    //float *m_row_pre = S + bc * br;
    //float *l_row_pre = m_row_pre + br;
    //float m_row_pre;
    //float l_row_pre;
    
    //TODO: take more thread 
 
    

    for(int j = 0; j < tc; j++)
    {
        //load k v to sram
        
        //sk[threadIdx.y * d + threadIdx.x] = K[qkv_offset + j * bc * d + threadIdx.y * d + threadIdx.x];
        //sv[threadIdx.y * d + threadIdx.x] = V[qkv_offset + j * bc * d + threadIdx.y * d + threadIdx.x];

        //update bc 
        if(j == tc - 1 && n % bc != 0)
        {
            bc = n % bc;
        } 
        


        for(int t =0; t < (blockDim.y + bc -1)/blockDim.y ; t++)
        {
            if(threadIdx.x < d && t * blockDim.y + threadIdx.y < bc && j * bc + t * blockDim.y + threadIdx.y < n)
            {
                sk[t * blockDim.y * d + threadIdx.y * d + threadIdx.x] = K[qkv_offset + j * bc * d + t * blockDim.y * d + threadIdx.y * d + threadIdx.x];
                sv[t * blockDim.y * d + threadIdx.y * d + threadIdx.x] = V[qkv_offset + j * bc * d + t * blockDim.y * d + threadIdx.y * d + threadIdx.x];
                //out_k[qkv_offset + j * bc * d + i * blockDim.y * d + threadIdx.y * d + threadIdx.x] = sk[i * blockDim.y * d + threadIdx.y * d + threadIdx.x]; 
            }
        }
        __syncthreads();
        //continue;
        for(int i = 0; i < tr ; i++)
        {
            if(i == tr - 1 && n % br != 0)
            {
                br = n % br;
            }
            //load q, o, l, m to sram
            //sq[threadIdx.y * d + threadIdx.x] = Q[qkv_offset + i * br * d + threadIdx.y * d + threadIdx.x];
            for(int t = 0; t < (blockDim.y + br - 1)/blockDim.y ; t++)
            {
                if(threadIdx.x < d && t * blockDim.y + threadIdx.y < br && i * br + t * blockDim.y + threadIdx.y < n)
                {
                    sq[t * blockDim.y * d + threadIdx.y * d + threadIdx.x] = Q[qkv_offset + i * br * d + t * blockDim.y * d + threadIdx.y * d + threadIdx.x];
                    //out_q[qkv_offset + i * br * d + t * blockDim.y * d + threadIdx.y * d + threadIdx.x] = sq[t * blockDim.y * d + threadIdx.y * d + threadIdx.x];
                }
            }

         
            __syncthreads();
            //continue;
            //compute score = sq * sk^T            
            for(int t = 0; t < (blockDim.y + br - 1)/blockDim.y ; t++)
            {   
                
                if(t * blockDim.y + threadIdx.y < br && i * br + t * blockDim.y + threadIdx.y < n)
                {
                    if(threadIdx.x < bc && j * bc + threadIdx.x < n)
                    {
                        float score = 0;
                        for(int k = 0; k < d; k++)
                        {
                            score += sq[t * blockDim.y * d + threadIdx.y * d + k] * sk[threadIdx.x * d + k];
                        }
                        score *= softmax_scale;
                        S[t * blockDim.y * bc + threadIdx.y * bc + threadIdx.x] = score;

                        //s[s_offset + i * br * n + (t * blockDim.y + threadIdx.y ) * n + j * bc + threadIdx.x] = score;
                    }
                }
                __syncthreads();
                float row_max;
                if(t * blockDim.y + threadIdx.y < br && i * br + t * blockDim.y + threadIdx.y < n)
                {
                    
                    row_max = S[t * blockDim.y * bc + threadIdx.y * bc];
                    for(int k = 1; k < bc && j * bc + k < n; k++)
                    {
                        if(S[t * blockDim.y * bc + threadIdx.y * bc + k] > row_max)
                        {
                            row_max = S[t * blockDim.y * bc + threadIdx.y * bc + k];
                        }
                    }
                }
                if(t * blockDim.y + threadIdx.y < br && i * br + t * blockDim.y + threadIdx.y < n)
                {
                    if(threadIdx.x < bc && j * bc + threadIdx.x < n)
                        S[t * blockDim.y * bc + threadIdx.y * bc + threadIdx.x] = __expf(S[t * blockDim.y * bc + threadIdx.y * bc + threadIdx.x] - row_max);
                }
                __syncthreads();
                if(t * blockDim.y + threadIdx.y < br && i * br + t * blockDim.y + threadIdx.y < n)  
                {
                    float row_sum = 0;  
                    for(int k = 0; k < bc && j * bc + k < n; k++)
                    {
                        //row_sum += S[threadIdx.y * bc + k];
                        row_sum += S[t * blockDim.y * bc + threadIdx.y * bc + k];
                    }

                    float m_row_pre = m[lm_offset + i * br + t * blockDim.y + threadIdx.y];
                    float l_row_pre = l[lm_offset + i * br + t * blockDim.y + threadIdx.y];
                    //update m_row_new, l_row_new
                    float m_row_new = max(m_row_pre, row_max);
                    float l_row_new = __expf(m_row_pre - m_row_new) * l_row_pre + __expf(row_max -m_row_new) * row_sum;
                    
                    if(threadIdx.x < d)
                    {
                        float pv = 0;
                        for(int k =0; k < bc && j * bc + k < n; k++)
                        {
                            //pv += S[threadIdx.y * bc + k] * sv[k * d + threadIdx.x];
                            pv += S[t * blockDim.y * bc + threadIdx.y * bc + k ] * sv[k * d + threadIdx.x];
                        }
                        O[qkv_offset + i * br * d + t * blockDim.y * d + threadIdx.y * d + threadIdx.x] = (1 / l_row_new) * (l_row_pre * __expf(m_row_pre - m_row_new) \
                            * O[qkv_offset + i * br *d + t * blockDim.y *d + threadIdx.y * d + threadIdx.x ] + __expf(row_max - m_row_new) * pv );
                        //out_o[qkv_offset + i * br * d + t * blockDim.y * d + threadIdx.y * d + threadIdx.x] = i * br * d + t * blockDim.y * d + threadIdx.y * d + threadIdx.x;
                        
                       
                    }
                    l[lm_offset + i * br + t * blockDim.y + threadIdx.y] = l_row_new;
                    m[lm_offset + i * br + t * blockDim.y + threadIdx.y] = m_row_new;
                   
                }                  

                
                __syncthreads();
            }
            __syncthreads();
           
            
               
        }
        
        __syncthreads();
    }


}

torch::Tensor forward(torch::Tensor q, torch::Tensor k, torch::Tensor v, const int max_thread_num)
{
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    //const int bc = max_thread_num / dim;
    //const int br = max_thread_num / dim;
    const int b = q.size(0);
    const int h = q.size(1);
    const int n = q.size(2);
    const int d = q.size(3);
    const int bc = ceil(float(max_sram_size) / sizeof(float) / (4 * d));
    const int br = min(bc, d);
    const int tc = ceil(float(n) / bc);
    const int tr = ceil(float(n) / br);
    const float softmax_scale = 1.0 / sqrt(d);
    //init o,l,m to hbm
    torch::Tensor o = torch::zeros_like(q);
    torch::Tensor l = torch::zeros({b, h, n});
    torch::Tensor m = torch::full({b, h, n}, -1e9);
    torch::Tensor s = torch::zeros({b, h, n, n});
    torch::Tensor out_q = torch::zeros_like(q);
    torch::Tensor out_k = torch::zeros_like(k);
    torch::Tensor out_o = torch::zeros_like(q);
    torch::Device device(torch::kCUDA);
    //o = o.to(device);
    l = l.to(device);
    m = m.to(device);
    s = s.to(device);
    printf("q: (%d %d %d %d)\n", q.size(0), q.size(1), q.size(2), q.size(3));
    const int sram_size = br * d * sizeof(float) + bc * d * 2 * sizeof(float) + bc * br * sizeof(float) + 2 * br * sizeof(float);
    printf("bc: %d, br: %d, tc: %d, tr: %d \n", bc, br, tc, tr);    
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);
    printf("Max elements: %d\n", max_sram_size / sizeof(float));
    printf("Max matrix size on shared memory:(%d %d)\n", max_sram_size / sizeof(float) / d, d);

    unsigned int block_d = max(d, bc);
    unsigned int block_h = max_thread_num / block_d;
    dim3 grid(h, b);
    printf("grid: (%d %d %d)\n", grid.x, grid.y, grid.z);
    dim3 block(block_d, block_h);
    printf("block: (%d %d)\n", block.x, block.y);


    flash_attn_kernel<<<grid, block, sram_size>>>(q.data_ptr<float>(), k.data_ptr<float>(), 
        v.data_ptr<float>(), o.data_ptr<float>(), bc, br, n, d, tc, tr, m.data_ptr<float>(),
        l.data_ptr<float>(), softmax_scale, s.data_ptr<float>(), out_q.data_ptr<float>(), out_k.data_ptr<float>(), out_o.data_ptr<float>());
    hipDeviceSynchronize();
    return o;
}

//main
int main(int argc, char **argv)
{
    torch::Tensor q = torch::randn({16, 12, 128, 32}).to(torch::kCUDA);
    torch::Tensor k = torch::randn({16, 12, 128, 32}).to(torch::kCUDA);
    torch::Tensor v = torch::randn({16, 12, 128, 32}).to(torch::kCUDA);
    torch::Tensor s = torch::zeros({16, 12, 128, 128}).to(torch::kCUDA);
    s = forward(q, k, v, 1024);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("Error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
    return 0;
}
